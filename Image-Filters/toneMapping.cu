#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "toneMapping.h"
#include "hip/hip_runtime.h"

size_t numBins = 1024;
#define BLOCK_SIZE 32
const dim3 blockSize(32, 16, 1);
const int numThreads = 192;

namespace toneMapping {
	void rgb_to_xyY(size_t rows, size_t cols, float *red, float *green, float *blue, float *x, float *y, float *logY) {
		float X, Y, Z, L;
		for (size_t i = 0; i < rows*cols; i++) {
				X = (red[i] * 0.4124f) + (green[i] * 0.3576f) + (blue[i] * 0.1805f);
				Y = (red[i] * 0.2126f) + (green[i] * 0.7152f) + (blue[i] * 0.0722f);
				Z = (red[i] * 0.0193f) + (green[i] * 0.1192f) + (blue[i] * 0.9505f);
				L = X + Y + Z;
				x[i] = X / L;
				y[i] = Y / L;
				logY[i] = log10f(0.0001 + Y);
		}
	}

	void calculateCDF(size_t rows, size_t cols, size_t numBins, float *luminance, unsigned int* cdf, float &min_logLum, float &max_logLum) {
		min_logLum = luminance[0];
		max_logLum = luminance[0];

		for (size_t i = 1; i < rows*cols; ++i) {
				min_logLum = std::min(luminance[i], min_logLum);
				max_logLum = std::max(luminance[i], max_logLum);
		}

		float logLumRange = max_logLum - min_logLum;

		unsigned int *histo = new unsigned int[numBins];

		for (size_t i = 0; i < numBins; ++i) histo[i] = 0;

		for (size_t i = 0; i < rows*cols; ++i) {
				unsigned int bin = std::min(static_cast<unsigned int>(numBins - 1),
					static_cast<unsigned int>((luminance[i] - min_logLum) / logLumRange * numBins));
				histo[bin]++;
		}

		cdf[0] = 0;
		for (size_t i = 1; i < numBins; ++i) {
			cdf[i] = cdf[i - 1] + histo[i - 1];
		}

		delete[] histo;
	}

	void mapImage(size_t rows, size_t cols, float min_logLum, float max_logLum, unsigned int* cdf, 
		float *x, float *y, float *logY, float *red, float *green, float *blue) {
		float *norm_cdf = (float *)malloc(sizeof(float)*numBins);

		const float normalization_constant = 1.f / cdf[numBins - 1];
		for (size_t i = 0; i < numBins; i++)
		{
			unsigned int input_value = cdf[i];
			float        output_value = (float)input_value * normalization_constant;
			norm_cdf[i] = output_value;
		}

		float log_Y_range = max_logLum - min_logLum;

		for (size_t i = 0; i < rows*cols; i++) {
				float temp_x = x[i];
				float temp_y = y[i];
				float temp_log_Y = logY[i];
				int   bin_index = std::min((int)numBins - 1, int((numBins * (temp_log_Y - min_logLum)) / log_Y_range));
				float Y_new = norm_cdf[bin_index];
				float X_new = temp_x * (Y_new / temp_y);
				float Z_new = (1 - temp_x - temp_y) * (Y_new / temp_y);

				float r_new = (X_new *  3.2406f) + (Y_new * -1.5372f) + (Z_new * -0.4986f);
				float g_new = (X_new * -0.9689f) + (Y_new *  1.8758f) + (Z_new *  0.0415f);
				float b_new = (X_new *  0.0557f) + (Y_new * -0.2040f) + (Z_new *  1.0570f);

				red[i] = r_new;
				green[i] = g_new;
				blue[i] = b_new;
			}

		delete[] norm_cdf;
	}

	int cpuMap(size_t rows, size_t cols, float *imgPtr) {
		size_t numPixels = rows * cols;
		float *red = new float[numPixels];
		float *green = new float[numPixels];
		float *blue = new float[numPixels];
		float *x = new float[numPixels];
		float *y = new float[numPixels];
		float *logY = new float[numPixels];

		for (size_t i = 0; i < numPixels; ++i) {
			blue[i] = imgPtr[3 * i + 0];
			green[i] = imgPtr[3 * i + 1];
			red[i] = imgPtr[3 * i + 2];
		}

		unsigned int *cdf = (unsigned int *)malloc(sizeof(unsigned int)*numBins);
		float min_logLum, max_logLum;
		rgb_to_xyY(rows, cols, red, green, blue, x, y, logY);
		/*std::cout << "CDF" << std::endl;
		for (size_t i = 0; i < numPixels; i++)
		{
			std::cout << i << ", " << logY[i] << std::endl;
		}*/
		calculateCDF(rows, cols, numBins, logY, cdf, min_logLum, max_logLum);
		mapImage(rows, cols, min_logLum, max_logLum, cdf, x, y, logY, red, green, blue);

		for (int i = 0; i < numPixels; ++i) {
			imgPtr[3 * i + 0] = blue[i];
			imgPtr[3 * i + 1] = green[i];
			imgPtr[3 * i + 2] = red[i];
		}
		delete[] cdf;
		delete[] red;
		delete[] green;
		delete[] blue;
		delete[] x;
		delete[] y;
		delete[] logY;
		return 1;
	}

	__global__ void rgb2xyY(float* red, float* green, float* blue, float* d_x, float* d_y,
		float* d_logY, int rows, int cols) {
		int  ny = rows;
		int  nx = cols;
		int2 image_index_2d = make_int2((blockIdx.x * blockDim.x) + threadIdx.x, (blockIdx.y * blockDim.y) + threadIdx.y);
		int  image_index_1d = (nx * image_index_2d.y) + image_index_2d.x;

		if (image_index_2d.x < nx && image_index_2d.y < ny)
		{
			float r = red[image_index_1d];
			float g = green[image_index_1d];
			float b = blue[image_index_1d];

			float X = (r * 0.4124f) + (g * 0.3576f) + (b * 0.1805f);
			float Y = (r * 0.2126f) + (g * 0.7152f) + (b * 0.0722f);
			float Z = (r * 0.0193f) + (g * 0.1192f) + (b * 0.9505f);

			float L = X + Y + Z;
			float x = X / L;
			float y = Y / L;

			float log_Y = log10f(0.0001f + Y);

			d_x[image_index_1d] = x;
			d_y[image_index_1d] = y;
			d_logY[image_index_1d] = log_Y;
		}
	}

	// calculate reduce max or min and stick the value in d_answer.
	__global__
		void reduce_minmax_kernel(const float* const d_in, float* d_out, const size_t size, int minmax) {
			extern __shared__ float shared[];

			int mid = threadIdx.x + blockDim.x * blockIdx.x;
			int tid = threadIdx.x;

			// we have 1 thread per block, so copying the entire block should work fine
			if (mid < size) {
				shared[tid] = d_in[mid];
			}
			else {
				if (minmax == 0)
					shared[tid] = FLT_MAX;
				else
					shared[tid] = -FLT_MAX;
			}

			// wait for all threads to copy the memory
			__syncthreads();

			// don't do any thing with memory if we happen to be far off ( I don't know how this works with
			// sync threads so I moved it after that point )
			if (mid >= size) {
				if (tid == 0) {
					if (minmax == 0)
						d_out[blockIdx.x] = FLT_MAX;
					else
						d_out[blockIdx.x] = -FLT_MAX;

				}
				return;
			}

			for (unsigned int s = blockDim.x / 2; s > 0; s /= 2) {
				if (tid < s) {
					if (minmax == 0) {
						shared[tid] = min(shared[tid], shared[tid + s]);
					}
					else {
						shared[tid] = max(shared[tid], shared[tid + s]);
					}
				}

				__syncthreads();
			}

			if (tid == 0) {
				d_out[blockIdx.x] = shared[0];
			}
		}

	__global__ void histogram_kernel(unsigned int* d_bins, const float* d_in, const int bin_count, const float lum_min, const float lum_max, const int size) {
		int mid = threadIdx.x + blockDim.x * blockIdx.x;
		if (mid >= size)
			return;
		float lum_range = lum_max - lum_min;
		int bin = ((d_in[mid] - lum_min) / lum_range) * bin_count;

		atomicAdd(&d_bins[bin], 1);
	}

	__global__ void scan_kernel(unsigned int* d_bins, int size) {
			int mid = threadIdx.x + blockDim.x * blockIdx.x;
			if (mid >= size)
				return;

			for (int s = 1; s <= size; s *= 2) {
				int spot = mid - s;

				unsigned int val = 0;
				if (spot >= 0)
					val = d_bins[spot];
				__syncthreads();
				if (spot >= 0)
					d_bins[mid] += val;
				__syncthreads();

			}
		}

	__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n)
	{
		const float normalization_constant = 1.f / d_input_cdf[n - 1];

		int global_index_1d = (blockIdx.x * blockDim.x) + threadIdx.x;

		if (global_index_1d < n)
		{
			unsigned int input_value = d_input_cdf[global_index_1d];
			float        output_value = input_value * normalization_constant;

			d_output_cdf[global_index_1d] = output_value;
		}
	}

	__global__ void tonemap(
		float* d_x,
		float* d_y,
		float* d_log_Y,
		float* d_cdf_norm,
		float* d_r_new,
		float* d_g_new,
		float* d_b_new,
		float  min_log_Y,
		float  max_log_Y,
		float  log_Y_range,
		int    num_bins,
		int    num_pixels_y,
		int    num_pixels_x)
	{
		int  ny = num_pixels_y;
		int  nx = num_pixels_x;
		int2 image_index_2d = make_int2((blockIdx.x * blockDim.x) + threadIdx.x, (blockIdx.y * blockDim.y) + threadIdx.y);
		int  image_index_1d = (nx * image_index_2d.y) + image_index_2d.x;

		if (image_index_2d.x < nx && image_index_2d.y < ny)
		{
			float x = d_x[image_index_1d];
			float y = d_y[image_index_1d];
			float log_Y = d_log_Y[image_index_1d];
			int   bin_index = min(num_bins - 1, int((num_bins * (log_Y - min_log_Y)) / log_Y_range));
			float Y_new = d_cdf_norm[bin_index];

			float X_new = x * (Y_new / y);
			float Z_new = (1 - x - y) * (Y_new / y);

			float r_new = (X_new *  3.2406f) + (Y_new * -1.5372f) + (Z_new * -0.4986f);
			float g_new = (X_new * -0.9689f) + (Y_new *  1.8758f) + (Z_new *  0.0415f);
			float b_new = (X_new *  0.0557f) + (Y_new * -0.2040f) + (Z_new *  1.0570f);

			d_r_new[image_index_1d] = r_new;
			d_g_new[image_index_1d] = g_new;
			d_b_new[image_index_1d] = b_new;
		}
	}

	float reduce_minmax(const float* const d_in, const size_t size, int minmax) {
		// we need to keep reducing until we get to the amount that we consider 
		// having the entire thing fit into one block size
		size_t curr_size = size;
		float* dev_data;

		hipMalloc(&dev_data, sizeof(float)* size);
		hipMemcpy(dev_data, d_in, sizeof(float)* size, hipMemcpyDeviceToDevice);


		float* dev_temp;

		dim3 thread_dim(BLOCK_SIZE);
		const int shared_mem_size = sizeof(float)*BLOCK_SIZE;
		int maxSize;
		while (1) {
			maxSize = (int)ceil((float)curr_size / (float)BLOCK_SIZE) + 1;
			hipMalloc(&dev_temp, sizeof(float)* maxSize);

			dim3 block_dim((int)ceil((float)size / (float)BLOCK_SIZE) + 1);
			reduce_minmax_kernel << <block_dim, thread_dim, shared_mem_size >> >(dev_data, dev_temp, curr_size, minmax);
			hipDeviceSynchronize();


			// move the current input to the output, and clear the last input if necessary
			hipFree(dev_data);
			dev_data = dev_temp;

			if (curr_size <  BLOCK_SIZE)
				break;

			curr_size = maxSize;
		}

		// theoretically we should be 
		float result;
		hipMemcpy(&result, dev_temp, sizeof(float), hipMemcpyDeviceToHost);
		hipFree(dev_temp);
		return result;
	}

	int gpuMap(size_t rows, size_t cols, float *imgPtr) {
		float *dev_red, *dev_green, *dev_blue, *dev_x, *dev_y, *dev_logY;
		size_t numPixels = rows * cols;
		hipMalloc((void**)&dev_red, sizeof(float)*numPixels);
		hipMalloc((void**)&dev_green, sizeof(float)*numPixels);
		hipMalloc((void**)&dev_blue, sizeof(float)*numPixels);
		hipMalloc((void**)&dev_x, sizeof(float)*numPixels);
		hipMalloc((void**)&dev_y, sizeof(float)*numPixels);
		hipMalloc((void**)&dev_logY, sizeof(float)*numPixels);

		float *red = new float[numPixels];
		float *green = new float[numPixels];
		float *blue = new float[numPixels];

		for (size_t i = 0; i < numPixels; ++i) {
			blue[i] = imgPtr[3 * i + 0];
			green[i] = imgPtr[3 * i + 1];
			red[i] = imgPtr[3 * i + 2];
		}

		hipMemcpy(dev_red, red, sizeof(float)*numPixels, hipMemcpyHostToDevice);
		hipMemcpy(dev_green, green, sizeof(float)*numPixels, hipMemcpyHostToDevice);
		hipMemcpy(dev_blue, blue, sizeof(float)*numPixels, hipMemcpyHostToDevice);

		const dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y, 1);
		rgb2xyY << <gridSize, blockSize >> >(dev_red, dev_green, dev_blue,	dev_x, dev_y, dev_logY, rows, cols);

		float *logY = (float *)malloc(sizeof(float)*numPixels);
		hipMemcpy(logY, dev_logY, numPixels*sizeof(float), hipMemcpyDeviceToHost);

		// Calculate CDF
		float min_logLum, max_logLum;
		unsigned int *dev_cdf;
		hipMalloc((void**)&dev_cdf, sizeof(unsigned int)*numBins);
		min_logLum = reduce_minmax(dev_logY, numPixels, 0);
		max_logLum = reduce_minmax(dev_logY, numPixels, 1);

		printf("got min of %f\n", min_logLum);
		printf("got max of %f\n", max_logLum);
		printf("numBins %d\n", numBins);

		unsigned int* dev_bins;

		hipMalloc(&dev_bins, sizeof(unsigned int)*numBins);
		hipMemset(dev_bins, 0, sizeof(unsigned int)*numBins);
		dim3 thread_dim(1024);
		dim3 hist_block_dim((int)ceil((float)numPixels / (float)thread_dim.x) + 1);
		histogram_kernel << <hist_block_dim, thread_dim >> >(dev_bins, dev_logY, numBins, min_logLum, max_logLum, numPixels);

		
		dim3 scan_block_dim((int)ceil((float)numBins / (float)thread_dim.x) + 1);

		scan_kernel << <scan_block_dim, thread_dim >> >(dev_bins, numBins);
		hipDeviceSynchronize();

		hipMemcpy(dev_cdf, dev_bins, sizeof(unsigned int)*numBins, hipMemcpyDeviceToDevice);
		hipFree(dev_bins);


		// Map
		float *dev_cdfNorm;
		hipMalloc(&dev_cdfNorm, sizeof(float)* numBins);

		normalize_cdf << < (numBins + numThreads - 1) / numThreads, numThreads >> >(dev_cdf, dev_cdfNorm, numBins);

		hipDeviceSynchronize();
		float log_Y_range = max_logLum - min_logLum;
		tonemap << <gridSize, blockSize >> >(dev_x, dev_y, dev_logY,
			dev_cdfNorm,
			dev_red, dev_green, dev_blue,
			min_logLum, max_logLum,
			log_Y_range, numBins,
			rows, cols);
		hipDeviceSynchronize();

		// Copy back data
		hipMemcpy(red, dev_red, sizeof(float)* numPixels, hipMemcpyDeviceToHost);
		hipMemcpy(green, dev_green, sizeof(float)* numPixels, hipMemcpyDeviceToHost);
		hipMemcpy(blue, dev_blue, sizeof(float)* numPixels, hipMemcpyDeviceToHost);

		for (int i = 0; i < numPixels; ++i) {
			imgPtr[3 * i + 0] = blue[i];
			imgPtr[3 * i + 1] = green[i];
			imgPtr[3 * i + 2] = red[i];
		}

		hipFree(dev_red);
		hipFree(dev_green);
		hipFree(dev_blue);
		hipFree(dev_x);
		hipFree(dev_y);
		hipFree(dev_logY);
		hipFree(dev_cdf);
		hipFree(dev_cdfNorm);

		delete[] red;
		delete[] green;
		delete[] blue;


		return 1;
	}
}
